#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

extern "C" void launch_vector_add(const float* A, const float* B, float* C,
                                  int N) {
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, N * sizeof(float));
  hipMalloc((void**)&d_B, N * sizeof(float));
  hipMalloc((void**)&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  vector_add<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);

  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
